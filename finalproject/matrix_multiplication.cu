﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <chrono>
#include <cstdlib> // For system()
#include <vector> // For dynamic arrays

// Helper function to check for CUDA errors
void checkCUDAError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << " Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Helper function to check for cuSPARSE errors
void checkCUSPARSEError(hipsparseStatus_t err, const char* msg) {
    if (err != HIPSPARSE_STATUS_SUCCESS) {
        std::cerr << msg << " Error: ";
        if (err == HIPSPARSE_STATUS_NOT_INITIALIZED) {
            std::cerr << "CUSPARSE_STATUS_NOT_INITIALIZED" << std::endl;
        }
        else if (err == HIPSPARSE_STATUS_ALLOC_FAILED) {
            std::cerr << "CUSPARSE_STATUS_ALLOC_FAILED" << std::endl;
        }
        else if (err == HIPSPARSE_STATUS_INVALID_VALUE) {
            std::cerr << "CUSPARSE_STATUS_INVALID_VALUE" << std::endl;
        }
        else {
            std::cerr << "Unknown error" << std::endl;
        }
        exit(EXIT_FAILURE);
    }
}

void runSparseMultiplication(int N, float density) {
    std::cout << "Running sparse matrix multiplication..." << std::endl;

    // Start power logging
    std::system("start /B nvidia-smi --query-gpu=power.draw --format=csv -l 1 > power_log.txt");

    // Initialize cuSPARSE
    hipsparseHandle_t handle;
    checkCUSPARSEError(hipsparseCreate(&handle), "Failed to create cuSPARSE handle");

    // Use vectors to dynamically allocate space for sparse matrix values
    std::vector<int> h_rowOffsets(N + 1, 0);
    std::vector<int> h_colIndices;
    std::vector<float> h_values;

    // Populate sparse matrix with random values
    srand(0);
    int nnz = 0;
    for (int i = 0; i < N; ++i) {
        h_rowOffsets[i] = nnz; // Start of row
        for (int j = 0; j < N; ++j) {
            if ((rand() / static_cast<float>(RAND_MAX)) < density) {
                h_values.push_back(static_cast<float>(rand()) / RAND_MAX); // Add value
                h_colIndices.push_back(j); // Add column index
                ++nnz; // Increment nnz
            }
        }
    }
    h_rowOffsets[N] = nnz; // End of last row

    // Allocate device memory for CSR format
    float* d_values;
    int* d_rowOffsets;
    int* d_colIndices;
    float* d_denseB, * d_denseC;

    checkCUDAError(hipMalloc(&d_values, nnz * sizeof(float)), "Failed to allocate device memory for values");
    checkCUDAError(hipMalloc(&d_rowOffsets, (N + 1) * sizeof(int)), "Failed to allocate device memory for row offsets");
    checkCUDAError(hipMalloc(&d_colIndices, nnz * sizeof(int)), "Failed to allocate device memory for column indices");
    checkCUDAError(hipMalloc(&d_denseB, N * N * sizeof(float)), "Failed to allocate device memory for B");
    checkCUDAError(hipMalloc(&d_denseC, N * N * sizeof(float)), "Failed to allocate device memory for C");

    // Copy CSR data to device
    checkCUDAError(hipMemcpy(d_rowOffsets, h_rowOffsets.data(), (N + 1) * sizeof(int), hipMemcpyHostToDevice), "Failed to copy row offsets to device");
    checkCUDAError(hipMemcpy(d_colIndices, h_colIndices.data(), nnz * sizeof(int), hipMemcpyHostToDevice), "Failed to copy column indices to device");
    checkCUDAError(hipMemcpy(d_values, h_values.data(), nnz * sizeof(float), hipMemcpyHostToDevice), "Failed to copy values to device");

    // Run sparse matrix multiplication using cuSPARSE SpMM
    float alpha = 1.0f, beta = 0.0f;
    hipsparseSpMatDescr_t spMatA;
    hipsparseDnMatDescr_t dnMatB, dnMatC;
    void* dBuffer = nullptr;
    size_t bufferSize = 0;

    checkCUSPARSEError(hipsparseCreateCsr(&spMatA, N, N, nnz, d_rowOffsets, d_colIndices, d_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F),
        "Failed to create sparse matrix descriptor");

    checkCUSPARSEError(hipsparseCreateDnMat(&dnMatB, N, N, N, d_denseB, HIP_R_32F, HIPSPARSE_ORDER_ROW),
        "Failed to create dense matrix descriptor B");
    checkCUSPARSEError(hipsparseCreateDnMat(&dnMatC, N, N, N, d_denseC, HIP_R_32F, HIPSPARSE_ORDER_ROW),
        "Failed to create dense matrix descriptor C");

    checkCUSPARSEError(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, spMatA, dnMatB, &beta, dnMatC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize),
        "Failed to calculate buffer size for SpMM");

    checkCUDAError(hipMalloc(&dBuffer, bufferSize), "Failed to allocate buffer for SpMM");

    // Measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    checkCUSPARSEError(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, spMatA, dnMatB, &beta, dnMatC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer),
        "Failed to perform SpMM");

    checkCUDAError(hipDeviceSynchronize(), "CUDA synchronization failed");

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    std::cout << "Sparse matrix multiplication completed in " << elapsed.count() << " ms" << std::endl;

    // Stop power logging
    std::system("taskkill /IM nvidia-smi.exe /F");

    // Cleanup
    hipsparseDestroySpMat(spMatA);
    hipsparseDestroyDnMat(dnMatB);
    hipsparseDestroyDnMat(dnMatC);
    hipFree(dBuffer);
    hipsparseDestroy(handle);
    hipFree(d_denseB);
    hipFree(d_denseC);
    hipFree(d_rowOffsets);
    hipFree(d_colIndices);
    hipFree(d_values);
}

int main() {
    const int N = 16384;    // Matrix size
    const float density = 0.9f; // 0.1f = 10% non-zero elements

    runSparseMultiplication(N, density);

    return 0;
}
